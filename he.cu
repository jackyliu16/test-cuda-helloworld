#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK_CUDA(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

__global__ void printHello()
{
    int index = threadIdx.x + blockIdx.x * blockDim.x; // 计算全局索引
    printf("hello world from GPU by thread:%d\n", index);
}

int main()
{
    hipSetDevice(0);
    printf("hello, world");
    dim3 grid_dim = {1, 1, 1};
    dim3 block_dim = {4, 1, 1};
    printHello<<<grid_dim, block_dim>>>();
    // CHECK_CUDA(cudaDeviceSynchronize());
    hipDeviceSynchronize();
    return 0;
}
